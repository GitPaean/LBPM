// Basic cuda functions callable from C/C++ code
#include <hip/hip_runtime.h>

extern "C" void AllocateDeviceMemory(void** address, size_t size){
       	hipMalloc(address,size);
	hipError_t err = hipGetLastError();
	if (hipSuccess != err){
	   printf("Error in cudaMalloc\n");
	}	
}

extern "C" void CopyToDevice(void* dest, void* source, size_t size){
	hipMemcpy(dest,source,size,hipMemcpyHostToDevice);
	hipError_t err = hipGetLastError();
	if (hipSuccess != err){
	   printf("Error in hipMemcpy host to device \n");
	}
}


extern "C" void CopyToHost(void* dest, void* source, size_t size){
	hipMemcpy(dest,source,size,hipMemcpyDeviceToHost);
	hipError_t err = hipGetLastError();
	if (hipSuccess != err){
	   printf("Error in hipMemcpy device to host \n");
	}
}

extern "C" void DeviceBarrier(){
	hipDeviceSynchronize();
}
