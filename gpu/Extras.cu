// Basic cuda functions callable from C/C++ code
#include <hip/hip_runtime.h>
#include <stdio.h>

extern "C" void ScaLBL_AllocateDeviceMemory(void** address, size_t size){
       	hipMalloc(address,size);
	hipError_t err = hipGetLastError();
	if (hipSuccess != err){
	   printf("Error in cudaMalloc: %s \n",hipGetErrorString(err));
	}	
}

extern "C" void ScaLBL_CopyToDevice(void* dest, const void* source, size_t size){
	hipMemcpy(dest,source,size,hipMemcpyHostToDevice);
	hipError_t err = hipGetLastError();
	if (hipSuccess != err){
	   printf("Error in hipMemcpy (host->device): %s \n",hipGetErrorString(err));
	}
}


extern "C" void ScaLBL_CopyToHost(void* dest, const void* source, size_t size){
	hipMemcpy(dest,source,size,hipMemcpyDeviceToHost);
	hipError_t err = hipGetLastError();
	if (hipSuccess != err){
	   printf("Error in hipMemcpy (device->host): %s \n",hipGetErrorString(err));
	}
}

extern "C" void ScaLBL_DeviceBarrier(){
	hipDeviceSynchronize();
}
