// Basic cuda functions callable from C/C++ code
#include <hip/hip_runtime.h>

extern "C" void AllocateDeviceMemory(void** address, size_t size){
	hipMalloc(address,size);
}

extern "C" void CopyToDevice(void* dest, void* source, size_t size){
	hipMemcpy(dest,source,size,hipMemcpyHostToDevice);
}


extern "C" void CopyToHost(void* dest, void* source, size_t size){
	hipMemcpy(dest,source,size,hipMemcpyDeviceToHost);
}

extern "C" void DeviceBarrier(){
	hipDeviceSynchronize();
}
