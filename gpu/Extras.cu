// Basic cuda functions callable from C/C++ code
#include <hip/hip_runtime.h>
#include <stdio.h>

extern "C" void AllocateDeviceMemory(void** address, size_t size){
       	hipMalloc(address,size);
	hipError_t err = hipGetLastError();
	if (hipSuccess != err){
	   printf("Error in cudaMalloc: %s \n",hipGetErrorString(err));
	}	
}

extern "C" void CopyToDevice(void* dest, void* source, size_t size){
	hipMemcpy(dest,source,size,hipMemcpyHostToDevice);
	hipError_t err = hipGetLastError();
	if (hipSuccess != err){
	   printf("Error in hipMemcpy (host->device): %s \n",hipGetErrorString(err));
	}
}


extern "C" void CopyToHost(void* dest, void* source, size_t size){
	hipMemcpy(dest,source,size,hipMemcpyDeviceToHost);
	hipError_t err = hipGetLastError();
	if (hipSuccess != err){
	   printf("Error in hipMemcpy (device->host): %s \n",hipGetErrorString(err));
	}
}

extern "C" void DeviceBarrier(){
	hipDeviceSynchronize();
}
